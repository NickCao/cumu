#include "hip/hip_runtime.h"
<<<<<<< HEAD
#include <iostream>
#include <hip/hip_runtime_api.h>

#define N 10000000

__global__ void vector_add(float *out, float *a, float *b, int n) {
    for(int i = 0; i < n; i++){
        out[i] = a[i] + b[i];
    }
}

int main(){
    float *a, *b, *out;

    // Allocate memory
    a   = (float*)malloc(sizeof(float) * N);
    b   = (float*)malloc(sizeof(float) * N);
    out = (float*)malloc(sizeof(float) * N);

    // Initialize array
    for(int i = 0; i < N; i++){
        a[i] = 1.0f; b[i] = 2.0f;
    }

    vector_add<<<1,1>>>(out, a, b, N);
    hipError_t err = hipGetLastError();
    std::cout << hipGetErrorString(err) << std::endl;
}
=======
#include <iostream>
#include <hip/hip_runtime_api.h>

#define N 10000000

__global__ void vector_add(float *out, float *a, float *b, int n) {
    for(int i = 0; i < n; i++){
        out[i] = a[i] + b[i];
    }
}

int main(){
    float *a, *b, *out;

    // Allocate memory on the host
    a   = (float*)malloc(sizeof(float) * N);
    b   = (float*)malloc(sizeof(float) * N);
    out = (float*)malloc(sizeof(float) * N);

    // Initialize arrays
    for(int i = 0; i < N; i++){
        a[i] = 1.0f; b[i] = 2.0f;
    }

    vector_add<<<1,1>>>(out, a, b, N);
    hipError_t err = hipGetLastError();


    std::cout << "=========================================================" << std::endl;

    std::cout << ">>> Print Test >>> " << "HelloWorld!" << std::endl;
    std::cout << ">>> hipGetLastError Test >>> " << hipGetLastError() << std::endl;
    std::cout << ">>> hipGetErrorString Test >>> " << hipGetErrorString(err) << std::endl;

    std::cout << "=========================================================" << std::endl;
}
>>>>>>> 1d9fa9cdb73d57785fe8516cb9527cb594a08b26
